#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <torch/extension.h>

#include "cuda_utils.h"

// input: new_xyz(b, m, 3) xyz(b, n, 3)
// output: idx(b, m, nsample)
template<typename scalar_t>
__global__ void query_ball_point_kernel(
    int b,
    int n,
    int m,
    float radius,
    int nsample,
    const scalar_t *__restrict__ new_xyz,
    const scalar_t *__restrict__ xyz,
    int *__restrict__ idx) {
  int batch_index = blockIdx.x;
  xyz += batch_index * n * 3;
  new_xyz += batch_index * m * 3;
  idx += m * nsample * batch_index;

  int index = threadIdx.x;
  int stride = blockDim.x;

  float radius2 = radius * radius;
  for (int j = index; j < m; j += stride) {
    scalar_t new_x = new_xyz[j * 3 + 0];
    scalar_t new_y = new_xyz[j * 3 + 1];
    scalar_t new_z = new_xyz[j * 3 + 2];
    for (int k = 0, cnt = 0; k < n && cnt < nsample; ++k) {
      scalar_t x = xyz[k * 3 + 0];
      scalar_t y = xyz[k * 3 + 1];
      scalar_t z = xyz[k * 3 + 2];
      scalar_t d2 = (new_x - x) * (new_x - x) + (new_y - y) * (new_y - y) +
                 (new_z - z) * (new_z - z);
      if (d2 < radius2) {
        if (cnt == 0) {
          for (int l = 0; l < nsample; ++l) {
            idx[j * nsample + l] = k;
          }
        }
        idx[j * nsample + cnt] = k;
        ++cnt;
      }
    }
  }
}

at::Tensor query_ball_point_kernel_wrapper(
    int b,
    int n,
    int m,
    float radius,
    int nsample,
    const at::Tensor new_xyz,
    const at::Tensor xyz) {
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  at::Tensor idx = torch::zeros(
      {new_xyz.size(0), new_xyz.size(1), nsample},
      at::device(new_xyz.device()).dtype(at::ScalarType::Int));

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    xyz.scalar_type(), "query_ball_cuda", ([&] {
      query_ball_point_kernel<scalar_t><<<b, opt_n_threads(m), 0, stream>>>(
        b,
        n,
        m,
        radius,
        nsample,
        new_xyz.data_ptr<scalar_t>(),
        xyz.data_ptr<scalar_t>(),
        idx.data_ptr<int>());
  }));
  CUDA_CHECK_ERRORS();
  return idx;
}
