#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <vector>
#include <torch/extension.h>

#include "cuda_utils.h"

// input: unknown(b, n, 3) known(b, m, 3)
// output: dist2(b, n, 3), idx(b, n, 3)
template<typename scalar_t>
__global__ void three_nn_kernel(int b, int n, int m,
                                const scalar_t *__restrict__ unknown,
                                const scalar_t *__restrict__ known,
                                scalar_t *__restrict__ dist2,
                                int *__restrict__ idx) {
  int batch_index = blockIdx.x;
  unknown += batch_index * n * 3;
  known += batch_index * m * 3;
  dist2 += batch_index * n * 3;
  idx += batch_index * n * 3;

  int index = threadIdx.x;
  int stride = blockDim.x;
  for (int j = index; j < n; j += stride) {
    scalar_t ux = unknown[j * 3 + 0];
    scalar_t uy = unknown[j * 3 + 1];
    scalar_t uz = unknown[j * 3 + 2];

    double best1 = 1e40, best2 = 1e40, best3 = 1e40;
    int besti1 = 0, besti2 = 0, besti3 = 0;
    for (int k = 0; k < m; ++k) {
      scalar_t x = known[k * 3 + 0];
      scalar_t y = known[k * 3 + 1];
      scalar_t z = known[k * 3 + 2];
      scalar_t d = (ux - x) * (ux - x) + (uy - y) * (uy - y) + (uz - z) * (uz - z);
      if (d < best1) {
        best3 = best2;
        besti3 = besti2;
        best2 = best1;
        besti2 = besti1;
        best1 = d;
        besti1 = k;
      } else if (d < best2) {
        best3 = best2;
        besti3 = besti2;
        best2 = d;
        besti2 = k;
      } else if (d < best3) {
        best3 = d;
        besti3 = k;
      }
    }
    dist2[j * 3 + 0] = best1;
    dist2[j * 3 + 1] = best2;
    dist2[j * 3 + 2] = best3;

    idx[j * 3 + 0] = besti1;
    idx[j * 3 + 1] = besti2;
    idx[j * 3 + 2] = besti3;
  }
}

std::vector<at::Tensor> three_nn_kernel_wrapper(
    int b,
    int n,
    int m,
    const at::Tensor unknown,
    const at::Tensor known) {
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  at::Tensor idx = torch::zeros(
    {unknown.size(0), unknown.size(1), 3},
    at::device(unknown.device()).dtype(at::ScalarType::Int));
  at::Tensor dist2 =
      torch::zeros({unknown.size(0), unknown.size(1), 3},
      at::device(unknown.device()).dtype(unknown.scalar_type()));

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    unknown.scalar_type(), "three_nn_kernel_cuda", ([&] {
      three_nn_kernel<scalar_t><<<b, opt_n_threads(n), 0, stream>>>(
          b,
          n,
          m,
          unknown.data_ptr<scalar_t>(),
          known.data_ptr<scalar_t>(),
          dist2.data_ptr<scalar_t>(),
          idx.data_ptr<int>());
  }));

  CUDA_CHECK_ERRORS();
  return {dist2, idx};
}

// input: points(b, c, m), idx(b, n, 3), weight(b, n, 3)
// output: out(b, c, n)
template<typename scalar_t>
__global__ void three_interpolate_kernel(
    int b,
    int c,
    int m,
    int n,
    const scalar_t *__restrict__ points,
    const int *__restrict__ idx,
    const scalar_t *__restrict__ weight,
    scalar_t *__restrict__ out) {
  int batch_index = blockIdx.x;
  points += batch_index * m * c;

  idx += batch_index * n * 3;
  weight += batch_index * n * 3;

  out += batch_index * n * c;

  const int index = threadIdx.y * blockDim.x + threadIdx.x;
  const int stride = blockDim.y * blockDim.x;
  for (int i = index; i < c * n; i += stride) {
    const int l = i / n;
    const int j = i % n;
    scalar_t w1 = weight[j * 3 + 0];
    scalar_t w2 = weight[j * 3 + 1];
    scalar_t w3 = weight[j * 3 + 2];

    int i1 = idx[j * 3 + 0];
    int i2 = idx[j * 3 + 1];
    int i3 = idx[j * 3 + 2];

    out[i] = points[l * m + i1] * w1 + points[l * m + i2] * w2 +
             points[l * m + i3] * w3;
  }
}

at::Tensor three_interpolate_kernel_wrapper(
    int b,
    int c,
    int m,
    int n,
    const at::Tensor points,
    const at::Tensor idx,
    const at::Tensor weight) {
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  at::Tensor out =
      torch::zeros({points.size(0), points.size(1), idx.size(1)},
                   at::device(points.device()).dtype(points.scalar_type()));

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      points.scalar_type(), "three_interpolate_cuda", ([&] {
        three_interpolate_kernel<scalar_t><<<b, opt_block_config(n, c), 0, stream>>>(
          b,
          c,
          m,
          n,
          points.data_ptr<scalar_t>(),
          idx.data_ptr<int>(),
          weight.data_ptr<scalar_t>(),
          out.data_ptr<scalar_t>());
        }));

  CUDA_CHECK_ERRORS();
  return out;
}

// input: grad_out(b, c, n), idx(b, n, 3), weight(b, n, 3)
// output: grad_points(b, c, m)

template<typename scalar_t>
__global__ void three_interpolate_grad_kernel(
    int b,
    int c,
    int n,
    int m,
    const scalar_t *__restrict__ grad_out,
    const int *__restrict__ idx,
    const scalar_t *__restrict__ weight,
    scalar_t *__restrict__ grad_points) {
  int batch_index = blockIdx.x;
  grad_out += batch_index * n * c;
  idx += batch_index * n * 3;
  weight += batch_index * n * 3;
  grad_points += batch_index * m * c;

  const int index = threadIdx.y * blockDim.x + threadIdx.x;
  const int stride = blockDim.y * blockDim.x;
  for (int i = index; i < c * n; i += stride) {
    const int l = i / n;
    const int j = i % n;
    scalar_t w1 = weight[j * 3 + 0];
    scalar_t w2 = weight[j * 3 + 1];
    scalar_t w3 = weight[j * 3 + 2];

    int i1 = idx[j * 3 + 0];
    int i2 = idx[j * 3 + 1];
    int i3 = idx[j * 3 + 2];

    atomicAdd(grad_points + l * m + i1, grad_out[i] * w1);
    atomicAdd(grad_points + l * m + i2, grad_out[i] * w2);
    atomicAdd(grad_points + l * m + i3, grad_out[i] * w3);
  }
}

at::Tensor three_interpolate_grad_kernel_wrapper(
    int b,
    int c,
    int n,
    int m,
    const at::Tensor grad_out,
    const at::Tensor idx,
    const at::Tensor weight) {
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  at::Tensor grad_points = torch::zeros(
      {grad_out.size(0), grad_out.size(1), m},
      at::device(grad_out.device()).dtype(grad_out.scalar_type()));
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      grad_out.scalar_type(), "three_interpolate_grad_cuda", ([&] {
        three_interpolate_grad_kernel<<<b, opt_block_config(n, c), 0, stream>>>(
            b,
            c,
            n,
            m,
            grad_out.data_ptr<scalar_t>(),
            idx.data_ptr<int>(),
            weight.data_ptr<scalar_t>(),
            grad_points.data_ptr<scalar_t>());
        }));
  CUDA_CHECK_ERRORS();
  return grad_points;
}
